#include "hip/hip_runtime.h"
#include "gradient.hpp"
#include "kernel_gradient.cuh"
#include "kernels_for_hybrid.cu"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <omp.h>

#define Nb_time_mes 10

void init_templates()
{
  
/*
  reduce_last_in_one_thread<double><<<1,1>>>(NULL,NULL,0);
  gradient_kernel_2_beta_with_INDEXING<double><<<1,1>>>(NULL,NULL,NULL,NULL,NULL,NULL,NULL,NULL,0);
  gradient_kernel_2_beta_with_INDEXING<float><<<1,1>>>(NULL,NULL,NULL,NULL,NULL,NULL,NULL,NULL,0);
*/
}

template <typename T> 
void gradient_L_2_beta(T* deriv_dev, int* taille_deriv, int product_taille_deriv, T* params_dev, int* taille_params, int product_taille_params, T* residual_dev, int* taille_residual, int product_residual, T* std_map_dev, int* taille_std_map, int product_std_map, int n_gauss, float* temps)
{
    hipEvent_t record_event[2];
    float time_msec[1];
    for (int i=0;i<2;i++){
        checkCudaErrors(hipEventCreate(record_event+i));   
    }
    
    checkCudaErrors(hipDeviceSynchronize());

   int* taille_params_dev = NULL;
   int* taille_deriv_dev = NULL;
   int* taille_residual_dev = NULL;
   int* taille_std_map_dev = NULL;

    checkCudaErrors(hipDeviceSynchronize());

   checkCudaErrors(hipMalloc(&taille_deriv_dev, 3*sizeof(int)));
   checkCudaErrors(hipMalloc(&taille_params_dev, 3*sizeof(int)));
   checkCudaErrors(hipMalloc(&taille_std_map_dev, 2*sizeof(int)));
   checkCudaErrors(hipMalloc(&taille_residual_dev, 3*sizeof(int)));
   checkCudaErrors(hipDeviceSynchronize());
   
   checkCudaErrors(hipMemcpy(taille_deriv_dev, taille_deriv, 3*sizeof(int), hipMemcpyHostToDevice));
   checkCudaErrors(hipMemcpy(taille_params_dev, taille_params, 3*sizeof(int), hipMemcpyHostToDevice));
   checkCudaErrors(hipMemcpy(taille_std_map_dev, taille_std_map,2*sizeof(int), hipMemcpyHostToDevice));
   checkCudaErrors(hipMemcpy(taille_residual_dev, taille_residual, 3*sizeof(int), hipMemcpyHostToDevice));

   dim3 Dg, Db;

    Db.x = BLOCK_SIZE_X; //x
    Db.y = BLOCK_SIZE_Y; //y
    Db.z = BLOCK_SIZE_Z; //gaussiennes
        //deriv      --> (3g,y,x)  --> (z,y,x)
        //params     --> (3g,y,x)  --> (z,y,x)
/*
    Dg.x = ceil(taille_deriv[2]/T(BLOCK_SIZE_X));
    Dg.y = ceil(taille_deriv[1]/T(BLOCK_SIZE_Y));
    Dg.z = ceil(taille_deriv[3]/T(BLOCK_SIZE_Z));
*/

    Dg.x = ceil(taille_deriv[2]/T(BLOCK_SIZE_X));
    Dg.y = ceil(taille_deriv[1]/T(BLOCK_SIZE_Y));
    Dg.z = ceil(taille_deriv[0]/T(BLOCK_SIZE_Z));

/*
    Dg.x = ceil(taille_deriv[2]/T(BLOCK_SIZE_X));
    Dg.y = ceil(taille_deriv[1]/T(BLOCK_SIZE_Y));
    Dg.z = ceil(taille_residual[0]/T(BLOCK_SIZE_Z));
*/
    // Record the start event
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipEventRecord(record_event[0], NULL));
    checkCudaErrors(hipDeviceSynchronize());

    compute_nabla_Q<T><<<Dg,Db>>>(deriv_dev, taille_deriv_dev, params_dev, taille_params_dev, residual_dev, taille_residual_dev, std_map_dev, taille_std_map_dev, n_gauss);
//  gradient_kernel_2_beta_with_INDEXING_over_v<<<Dg,Db>>>(deriv_dev, taille_deriv_dev, params_dev, taille_params_dev, residual_dev, taille_residual_dev, std_map_dev, taille_std_map_dev, n_gauss);
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipEventRecord(record_event[1], NULL));
    checkCudaErrors(hipDeviceSynchronize());

    checkCudaErrors(hipEventSynchronize(record_event[1]));    
    for(int i = 0; i<2-1; i++){
      checkCudaErrors(hipEventElapsedTime(time_msec+i, record_event[i], record_event[i+1]));
    }
/*
    for (int i=0;i<2;i++){
        checkCudaErrors(hipEventDestroy(record_event+i));   
    }
*/

    temps[0] += time_msec[0];

  checkCudaErrors(hipFree(taille_deriv_dev));
  checkCudaErrors(hipFree(taille_params_dev));
  checkCudaErrors(hipFree(taille_std_map_dev));
  checkCudaErrors(hipFree(taille_residual_dev));
}













//                        map_norm_dev        d_array_f
template <typename T> 
void reduction_loop(T* array_in, T* d_array_f, int size_array){
    int N = ceil(log(T(size_array))/log(T(BLOCK_SIZE_REDUCTION)));
//    printf("N = %d\n", N);

    int GRID_SIZE_REDUCTION = int(ceil(T(size_array)/T(BLOCK_SIZE_REDUCTION)));
    int size_array_out_kernel = ceil(T(size_array)/T(BLOCK_SIZE_REDUCTION));
    checkCudaErrors(hipDeviceSynchronize());
    T* array_out_kernel=NULL;
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipMalloc(&array_out_kernel, size_array_out_kernel*sizeof(T)));    
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipMemset(array_out_kernel, 0., size_array_out_kernel*sizeof(T)));
    checkCudaErrors(hipDeviceSynchronize());

    bool reduction_in_one_thread = false;
    if(reduction_in_one_thread){
      reduce_last_in_one_thread<T><<<1,1>>>(array_in, d_array_f, size_array);
    }else{
      sum_reduction<T><<< GRID_SIZE_REDUCTION, BLOCK_SIZE_REDUCTION >>>(array_in, array_out_kernel, size_array);
      checkCudaErrors(hipDeviceSynchronize());
      if(size_array_out_kernel>1){
        reduce_last_in_one_thread<T><<<1,1>>>(array_out_kernel, d_array_f, size_array_out_kernel);
      }else{
        cpy_first_num_dev<T><<<1,1>>>( array_out_kernel, d_array_f);
      }
    }
  checkCudaErrors(hipDeviceSynchronize());
  checkCudaErrors(hipFree(array_out_kernel));

}	

template <typename T> 
void reduction_loop_save(T* array_in, T* d_array_f, int size_array){
    int GRID_SIZE_REDUCTION = int(ceil(T(size_array)/T(BLOCK_SIZE_REDUCTION)));
    int N = ceil(log(T(size_array))/log(T(BLOCK_SIZE_REDUCTION)));
    printf("N = %d\n", N);
    
    reduce_last_in_one_thread<T><<<1,1>>>(array_in, d_array_f, size_array);
    
/*
    checkCudaErrors(hipDeviceSynchronize());

    if (N==1){
        int size_array_out_kernel = ceil(T(size_array)/T(BLOCK_SIZE_REDUCTION));
        int copy_dev_blocks = ceil(T(size_array_out_kernel)/T(BLOCK_SIZE_REDUCTION));
        T* array_out_kernel=NULL;
        checkCudaErrors(hipMalloc(&array_out_kernel, size_array_out_kernel*sizeof(T)));

        checkCudaErrors(hipDeviceSynchronize());

        sum_reduction<T><<< GRID_SIZE_REDUCTION, BLOCK_SIZE_REDUCTION >>>(array_in, array_out_kernel, size_array);

        checkCudaErrors(hipDeviceSynchronize());

        cpy_first_num_dev<T><<<1,1>>>( array_out_kernel, d_array_f);
        hipFree(array_out_kernel);

    } else{
        int size_array_out_kernel = ceil(T(size_array)/T(BLOCK_SIZE_REDUCTION));
        T* array_out_kernel=NULL;
        checkCudaErrors(hipMalloc(&array_out_kernel, size_array_out_kernel*sizeof(T)));

        sum_reduction<T><<< GRID_SIZE_REDUCTION, BLOCK_SIZE_REDUCTION >>>(array_in, array_out_kernel, size_array);

        checkCudaErrors(hipDeviceSynchronize());

        T* array_in_copied_2;
        checkCudaErrors(hipMalloc(&array_in_copied_2, size_array_out_kernel*sizeof(T)));

        int copy_dev_blocks = ceil(T(size_array_out_kernel)/T(BLOCK_SIZE_REDUCTION));
        copy_dev<T><<< copy_dev_blocks , BLOCK_SIZE_REDUCTION >>>(array_out_kernel, array_in_copied_2, size_array_out_kernel);

        hipFree(array_out_kernel);

        checkCudaErrors(hipDeviceSynchronize());

        T size_array_out_kernel_2 = ceil(T(size_array)/T(pow(BLOCK_SIZE_REDUCTION,2)));
        T* array_out_kernel_2=NULL;
        checkCudaErrors(hipMalloc(&array_out_kernel_2, size_array_out_kernel_2*sizeof(T)));

        sum_reduction<T><<< GRID_SIZE_REDUCTION, BLOCK_SIZE_REDUCTION >>>(array_in_copied_2, array_out_kernel_2, size_array_out_kernel);

        checkCudaErrors(hipDeviceSynchronize());

        if(N>2){
        reduce_last_in_one_thread<T><<<1,1>>>(array_out_kernel_2, d_array_f, size_array_out_kernel_2);
        }
        else{
        cpy_first_num_dev<T><<<1,1>>>( array_out_kernel_2, d_array_f);
        }
        hipFree(array_in_copied_2);
        hipFree(array_out_kernel_2);
    }
  */
}	






template <typename T> 
T compute_residual_and_f(T* beta, int* taille_beta, int product_taille_beta, T* cube, int* taille_cube, int product_taille_cube, T* residual, int* taille_residual, int product_taille_residual, T* std_map, int* taille_std_map, int product_taille_std_map, int indice_x, int indice_y, int indice_v, int n_gauss)
{
    T* beta_dev = NULL;
    T* cube_dev = NULL;
    T* residual_dev = NULL;
    T* std_map_dev = NULL;

    checkCudaErrors(hipMalloc(&beta_dev, product_taille_beta*sizeof(T)));
    checkCudaErrors(hipMalloc(&residual_dev, product_taille_residual*sizeof(T)));
    checkCudaErrors(hipMalloc(&cube_dev, product_taille_cube*sizeof(T)));
    checkCudaErrors(hipMalloc(&std_map_dev, product_taille_std_map*sizeof(T)));

    checkCudaErrors(hipMemcpy(beta_dev, beta, product_taille_beta*sizeof(T), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(residual_dev, residual, product_taille_residual*sizeof(T), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(cube_dev, cube, product_taille_cube*sizeof(T), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(std_map_dev, std_map, product_taille_std_map*sizeof(T), hipMemcpyHostToDevice));
    
    dim3 Dg, Db;
    Db.x = BLOCK_SIZE_X_BIS; //
    Db.y = BLOCK_SIZE_Y_BIS; //
    Db.z = BLOCK_SIZE_Z_BIS; //

    Dg.x = ceil(T(indice_x)/T(BLOCK_SIZE_X_BIS));
    Dg.y = ceil(T(indice_y)/T(BLOCK_SIZE_Y_BIS));
    Dg.z = ceil(T(indice_v)/T(BLOCK_SIZE_Z_BIS));

/*
    T* cube_reconstructed = NULL;
    checkCudaErrors(hipMalloc(&cube_reconstructed, indice_x*indice_y*indice_v*sizeof(cube_reconstructed[0])));
  	checkCudaErrors(hipMemset(cube_reconstructed, 0., indice_x*indice_y*indice_v*sizeof(cube_reconstructed[0])));
    checkCudaErrors(hipDeviceSynchronize());
    kernel_hypercube_reconstructed<T><<<Dg,Db>>>(beta_dev, cube_reconstructed, indice_x, indice_y, indice_v, n_gauss);
    checkCudaErrors(hipDeviceSynchronize());
    kernel_residual_simple_difference<T><<<Dg,Db>>>(cube_dev, cube_reconstructed, residual_dev, indice_x, indice_y, indice_v);
    checkCudaErrors(hipFree(cube_reconstructed));
*/

    kernel_residual<T><<<Dg,Db>>>(beta_dev, cube_dev, residual_dev,indice_x, indice_y, indice_v, n_gauss);

    checkCudaErrors(hipMemcpy(residual, residual_dev, product_taille_residual*sizeof(T), hipMemcpyDeviceToHost));

    dim3 Dg_L2, Db_L2;
    Db_L2.x = BLOCK_SIZE_L2_X;
    Db_L2.y = BLOCK_SIZE_L2_Y;
    Db_L2.z = 1;
    Dg_L2.x = ceil(indice_x/T(BLOCK_SIZE_L2_X));
    Dg_L2.y = ceil(indice_y/T(BLOCK_SIZE_L2_Y));
    Dg_L2.z = 1;


    T* map_norm_dev = NULL;
    checkCudaErrors(hipMalloc(&map_norm_dev, indice_x*indice_y*sizeof(T)));

    compute_Q_map<T><<<Dg_L2, Db_L2>>>(map_norm_dev, residual_dev, std_map_dev, indice_x, indice_y, indice_v);

    T* d_array_f=NULL;
    checkCudaErrors(hipMalloc(&d_array_f, 1*sizeof(T))); // ERREUR ICI

    reduction_loop<T>(map_norm_dev, d_array_f, indice_x*indice_y);

    T* array_f = (T*)malloc(1*sizeof(T));

    checkCudaErrors(hipMemcpy(array_f, d_array_f, 1*sizeof(T), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(residual, residual_dev, product_taille_residual*sizeof(T), hipMemcpyDeviceToHost));
    T sum1 = array_f[0];
    free(array_f);

    checkCudaErrors(hipFree(d_array_f));
    checkCudaErrors(hipFree(map_norm_dev));
    checkCudaErrors(hipFree(beta_dev));
    checkCudaErrors(hipFree(cube_dev));
    checkCudaErrors(hipFree(std_map_dev));
    checkCudaErrors(hipFree(residual_dev));

    return sum1;
  }


template <typename T> 
T compute_residual_and_f_less_memory(T* beta_dev, int* taille_beta, int product_taille_beta, T* cube, int* taille_cube, int product_taille_cube, T* residual_dev, int* taille_residual, int product_taille_residual, T* std_map_dev, int* taille_std_map, int product_taille_std_map, int dim_x, int dim_y, int dim_v, int n_gauss, float* temps)
{

  hipEvent_t start, stop;
  checkCudaErrors(hipEventCreate(&start));
  checkCudaErrors(hipEventCreate(&stop));

  hipEventRecord(start);

    hipEvent_t record_event[4];
    for (int i=0;i<4;i++){
        checkCudaErrors(hipEventCreate(record_event+i));   
    }

    
    dim3 Dg, Db;
    Db.x = BLOCK_SIZE_X_BIS; //
    Db.y = BLOCK_SIZE_Y_BIS; //
    Db.z = BLOCK_SIZE_Z_BIS; //

    Dg.x = ceil(T(dim_x)/T(BLOCK_SIZE_X_BIS));
    Dg.y = ceil(T(dim_y)/T(BLOCK_SIZE_Y_BIS));
    Dg.z = ceil(T(dim_v)/T(BLOCK_SIZE_Z_BIS));

    // Record the start event
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipEventRecord(record_event[0], NULL));
    checkCudaErrors(hipDeviceSynchronize());

    //residual_dev was initialized with cube_flattened, we do : residual_dev = model - cube_flattened = model - residual_dev
    compute_residual<T><<<Dg,Db>>>(beta_dev, residual_dev,dim_x, dim_y, dim_v, n_gauss); 

//    test modulo optim
//    Dg.z = ceil(T(n_gauss*dim_v)/T(BLOCK_SIZE_Z_BIS));
//    checkCudaErrors(hipDeviceSynchronize());
//    compute_residual_modulo<T><<<Dg,Db>>>(beta_dev, residual_dev, dim_x, dim_y, n_gauss*dim_v, n_gauss); 


    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipEventRecord(record_event[1], NULL));
    checkCudaErrors(hipDeviceSynchronize());


    dim3 Dg_L2, Db_L2;
    Db_L2.x = BLOCK_SIZE_L2_X;
    Db_L2.y = BLOCK_SIZE_L2_Y;
    Db_L2.z = 1;
    Dg_L2.x = ceil(dim_x/T(BLOCK_SIZE_L2_X));
    Dg_L2.y = ceil(dim_y/T(BLOCK_SIZE_L2_Y));
    Dg_L2.z = 1;


    T* map_norm_dev = NULL;
    checkCudaErrors(hipMalloc(&map_norm_dev, dim_x*dim_y*sizeof(T)));

    compute_Q_map<T><<<Dg_L2, Db_L2>>>(map_norm_dev, residual_dev, std_map_dev, dim_x, dim_y, dim_v);



    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipEventRecord(record_event[2], NULL));
    checkCudaErrors(hipDeviceSynchronize());


    T* d_array_f=NULL;
    checkCudaErrors(hipMalloc(&d_array_f, 1*sizeof(T))); // ERREUR ICI

    reduction_loop<T>(map_norm_dev, d_array_f, dim_x*dim_y);

    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipEventRecord(record_event[3], NULL));
    checkCudaErrors(hipDeviceSynchronize());
    
    checkCudaErrors(hipEventSynchronize(record_event[1]));    
    checkCudaErrors(hipEventSynchronize(record_event[2]));
    checkCudaErrors(hipEventSynchronize(record_event[3]));

    for(int i = 0; i<3; i++){
      checkCudaErrors(hipEventElapsedTime(temps+i, record_event[i], record_event[i+1]));
    }
/*
    for (int i=0;i<4;i++){
        checkCudaErrors(hipEventDestroy(record_event+i));   
    }
*/
//    checkCudaErrors(hipEventDestroy(record_event));

    T* array_f = (T*)malloc(1*sizeof(T));

    checkCudaErrors(hipMemcpy(array_f, d_array_f, 1*sizeof(T), hipMemcpyDeviceToHost));
    T sum1 = array_f[0];
    free(array_f);

    checkCudaErrors(hipFree(d_array_f));
    checkCudaErrors(hipFree(map_norm_dev));

  checkCudaErrors(hipDeviceSynchronize());
  checkCudaErrors(hipEventRecord(stop));
  float milliseconds = 0;
  checkCudaErrors(hipDeviceSynchronize());
  checkCudaErrors(hipEventElapsedTime(&milliseconds, start, stop));
//	printf("gpu : %.16f\n",milliseconds/1000);//temps_kernel[0]+temps_kernel[1]+temps_kernel[2]);

    return sum1;
  }


template <typename T> 
void regularization(T* beta_dev, T* g_dev, T* b_params, T &f, int dim_x, int dim_y, int dim_v, parameters<T> &M, float* temps_kernel_regu){
//  printf("début f = %.26f\n",f);

  hipEvent_t start, stop;
  checkCudaErrors(hipEventCreate(&start));
  checkCudaErrors(hipEventCreate(&stop));

  hipEventRecord(start);

	int n_beta = (3*M.n_gauss*dim_x*dim_y)+M.n_gauss;
  T* b_params_dev = NULL;
  checkCudaErrors(hipMalloc(&b_params_dev, M.n_gauss*sizeof(T)));
  checkCudaErrors(hipMemcpy(b_params_dev, b_params, M.n_gauss*sizeof(T), hipMemcpyHostToDevice));

  T* array_f_dev = NULL;
  checkCudaErrors(hipMalloc(&array_f_dev, 1*sizeof(T)));
  checkCudaErrors(hipMemset(array_f_dev, 0., 1*sizeof(T)));

  T* array_f = NULL;
  array_f = (T*)malloc(1*sizeof(T));
  array_f[0]=0.;
//  cpy_first_num_dev<T><<<1,1>>>(array_f array_f_dev);

//  checkCudaErrors(hipMemcpy(array_f_dev, array_f, 1*sizeof(T), hipMemcpyHostToDevice));

  T* d_IMAGE_amp = NULL;
  checkCudaErrors(hipMalloc(&d_IMAGE_amp, dim_x*dim_y*sizeof(T)));
  checkCudaErrors(hipMemset(d_IMAGE_amp, 0., dim_x*dim_y*sizeof(T)));
  T* d_IMAGE_mu = NULL;
  checkCudaErrors(hipMalloc(&d_IMAGE_mu, dim_x*dim_y*sizeof(T)));
  checkCudaErrors(hipMemset(d_IMAGE_mu, 0., dim_x*dim_y*sizeof(T)));
  T* d_IMAGE_sig = NULL;
  checkCudaErrors(hipMalloc(&d_IMAGE_sig, dim_x*dim_y*sizeof(T)));
  checkCudaErrors(hipMemset(d_IMAGE_sig, 0., dim_x*dim_y*sizeof(T)));

  T* d_CONV_amp = NULL;
  checkCudaErrors(hipMalloc(&d_CONV_amp, (dim_x+4)*(dim_y+4)*sizeof(T)));
  T* d_CONV_mu = NULL;
  checkCudaErrors(hipMalloc(&d_CONV_mu, (dim_x+4)*(dim_y+4)*sizeof(T)));
  T* d_CONV_sig = NULL;
  checkCudaErrors(hipMalloc(&d_CONV_sig, (dim_x+4)*(dim_y+4)*sizeof(T)));
  T* d_CONV_CONV_amp = NULL;
  checkCudaErrors(hipMalloc(&d_CONV_CONV_amp, (dim_x+4)*(dim_y+4)*sizeof(T)));
  T* d_CONV_CONV_mu = NULL;
  checkCudaErrors(hipMalloc(&d_CONV_CONV_mu, (dim_x+4)*(dim_y+4)*sizeof(T)));
  T* d_CONV_CONV_sig = NULL;
  checkCudaErrors(hipMalloc(&d_CONV_CONV_sig, (dim_x+4)*(dim_y+4)*sizeof(T)));

  checkCudaErrors(hipDeviceSynchronize());
  checkCudaErrors(hipEventRecord(stop));
  float tmp = 0.;
  checkCudaErrors(hipDeviceSynchronize());
  checkCudaErrors(hipEventElapsedTime(&tmp, start, stop));
  checkCudaErrors(hipDeviceSynchronize());
  temps_kernel_regu[8]+=tmp; //reduction

  dim3 Dg_2D, Db_2D;
  Db_2D.x = BLOCK_SIZE_L2_X;
  Db_2D.y = BLOCK_SIZE_L2_Y;
  Db_2D.z = 1;
  Dg_2D.x = ceil(dim_x/T(BLOCK_SIZE_L2_X));
  Dg_2D.y = ceil(dim_y/T(BLOCK_SIZE_L2_Y));
  Dg_2D.z = 1;

  checkCudaErrors(hipDeviceSynchronize());

  for(int k = 0; k<M.n_gauss; k++){
    hipEvent_t record_event[8];
    float time_msec[7] = {0.,0.,0.,0.,0.,0.,0.};
    for (int i=0;i<8;i++){
        checkCudaErrors(hipEventCreate(record_event+i));   
    }
    // Record the start event
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipEventRecord(record_event[0], NULL));
    checkCudaErrors(hipDeviceSynchronize());

    get_gaussian_parameter_maps<T><<<Dg_2D, Db_2D>>>(beta_dev, d_IMAGE_amp, d_IMAGE_mu, d_IMAGE_sig, dim_x, dim_y, k);

    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipEventRecord(record_event[1], NULL));

    checkCudaErrors(hipMemset(d_CONV_amp, 0., (dim_x+4)*(dim_y+4)*sizeof(T)));
    checkCudaErrors(hipMemset(d_CONV_mu, 0., (dim_x+4)*(dim_y+4)*sizeof(T)));
    checkCudaErrors(hipMemset(d_CONV_sig, 0., (dim_x+4)*(dim_y+4)*sizeof(T)));

    checkCudaErrors(hipMemset(d_CONV_CONV_amp, 0., (dim_x+4)*(dim_y+4)*sizeof(T)));
    checkCudaErrors(hipMemset(d_CONV_CONV_mu, 0., (dim_x+4)*(dim_y+4)*sizeof(T)));
    checkCudaErrors(hipMemset(d_CONV_CONV_sig, 0., (dim_x+4)*(dim_y+4)*sizeof(T)));

    //Sets d_EXT_* arrays to 0.


    //Does the convolutions for each gaussian parameters of the k-th gaussian index
    float tmp_temps_mirror_and_conv[6] = {0.,0.,0.,0.,0.,0.};
    conv_twice_and_copy<T>(d_IMAGE_amp, d_CONV_amp, d_CONV_CONV_amp, dim_x, dim_y,tmp_temps_mirror_and_conv);
    conv_twice_and_copy<T>(d_IMAGE_mu, d_CONV_mu, d_CONV_CONV_mu, dim_x, dim_y,tmp_temps_mirror_and_conv);
    conv_twice_and_copy<T>(d_IMAGE_sig, d_CONV_sig, d_CONV_CONV_sig, dim_x, dim_y,tmp_temps_mirror_and_conv);



    float tmp_temps_R[2] = {0.,0.};
    update_array_f_dev_sort_fast<T>(M.lambda_amp, M.lambda_mu, M.lambda_sig, M.lambda_var_sig, array_f_dev, d_CONV_amp, d_CONV_mu, d_CONV_sig, d_IMAGE_sig, dim_x, dim_y, k, b_params_dev,tmp_temps_R);
//    display_size<<<1,1>>>(array_f_dev, 1);

//    display_size<<<1,1>>>(array_f_dev, 1);
//    display_size<<<1,1>>>(b_params_dev, M.n_gauss);
//    exit(0);

    dim3 Dg_ud, Db_ud;
    Db_ud.x = BLOCK_SIZE_L2_X;
    Db_ud.y = BLOCK_SIZE_L2_Y;
    Db_ud.z = 1;
    Dg_ud.x = ceil(T(dim_x)/T(BLOCK_SIZE_L2_X));
    Dg_ud.y = ceil(T(dim_y)/T(BLOCK_SIZE_L2_Y));
    Dg_ud.z = 1;

    checkCudaErrors(hipDeviceSynchronize());

    double temps_test__ = omp_get_wtime();

    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipEventRecord(record_event[2], NULL));
    checkCudaErrors(hipDeviceSynchronize());

    compute_nabla_R_wrt_theta<T><<<Dg_ud,Db_ud>>>(g_dev, M.lambda_amp, M.lambda_mu, M.lambda_sig, M.lambda_var_sig, d_CONV_CONV_amp, d_CONV_CONV_mu, d_CONV_CONV_sig, d_IMAGE_sig, b_params_dev, dim_y, dim_x, k);

    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipEventRecord(record_event[3], NULL));
    checkCudaErrors(hipDeviceSynchronize());

    temps_test += omp_get_wtime() - temps_test__;
    

    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipEventRecord(record_event[4], NULL));
    checkCudaErrors(hipDeviceSynchronize());

    T* d_image_sigma_reduc = NULL;
    checkCudaErrors(hipMalloc(&d_image_sigma_reduc, 1*sizeof(T)));
  	checkCudaErrors(hipMemset(d_image_sigma_reduc, 0., 1*sizeof(T)));

    reduction_loop<T>(d_IMAGE_sig, d_image_sigma_reduc, dim_y*dim_x);

    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipEventRecord(record_event[5], NULL));
    checkCudaErrors(hipDeviceSynchronize());


    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipEventRecord(record_event[6], NULL));
    checkCudaErrors(hipDeviceSynchronize());

//    kernel_conv_g_reduction_sort<T><<<1,1>>>(n_beta+1, d_g, d_image_sigma_reduc, M.lambda_var_sig, M.n_gauss, b_params_dev, k, dim_x, dim_y);
    compute_nabla_R_wrt_m<T><<<1,1>>>(n_beta+1, g_dev, d_image_sigma_reduc, M.lambda_var_sig, M.n_gauss, b_params_dev, k, dim_x, dim_y);

    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipFree(d_image_sigma_reduc));

    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipEventRecord(record_event[7], NULL));
    
    checkCudaErrors(hipEventSynchronize(record_event[1]));    
    checkCudaErrors(hipEventSynchronize(record_event[2]));
    checkCudaErrors(hipEventSynchronize(record_event[3]));
    checkCudaErrors(hipEventSynchronize(record_event[4]));
    checkCudaErrors(hipEventSynchronize(record_event[5]));
    checkCudaErrors(hipEventSynchronize(record_event[6]));
    checkCudaErrors(hipEventSynchronize(record_event[7]));

    for(int i = 0; i<8-1; i++){
      checkCudaErrors(hipEventElapsedTime(time_msec+i, record_event[i], record_event[i+1]));
    }

    temps_kernel_regu[0]+=time_msec[0]; //get_gaussian_parameter_maps
    temps_kernel_regu[1]+=tmp_temps_mirror_and_conv[0]+tmp_temps_mirror_and_conv[3]; //perform_mirror_effect_before_convolution
    temps_kernel_regu[2]+=time_msec[4]+tmp_temps_mirror_and_conv[1]+tmp_temps_mirror_and_conv[4]; //ConvKernel
    temps_kernel_regu[3]+=tmp_temps_mirror_and_conv[2]+tmp_temps_mirror_and_conv[5]; //copy_gpu
    temps_kernel_regu[4]+=tmp_temps_R[0]; //compute_R_map
    temps_kernel_regu[5]+=time_msec[2]; //compute_nabla_R_wrt_theta
    temps_kernel_regu[6]+=time_msec[6]; //compute_nabla_R_wrt_m
    temps_kernel_regu[7]+=time_msec[4]+tmp_temps_R[1]; //reduction

    checkCudaErrors(hipDeviceSynchronize());

//init_extended_array_sort(T* d_IMAGE_amp, T* d_EXT_amp, int dim_x, int dim_y){

  }


//  display_size<<<1,1>>>(d_g, n_beta);

  checkCudaErrors(hipDeviceSynchronize());
//  exit(0);

  checkCudaErrors(hipDeviceSynchronize());

  checkCudaErrors(hipMemcpy(array_f, array_f_dev, 1*sizeof(T), hipMemcpyDeviceToHost));
  f += array_f[0];
//  printf("f = %.26f\n",f);
//	std::cin.ignore();

  checkCudaErrors(hipFree(array_f_dev));
  free(array_f);

  checkCudaErrors(hipFree(b_params_dev));
  checkCudaErrors(hipFree(d_IMAGE_amp));
  checkCudaErrors(hipFree(d_IMAGE_mu));
  checkCudaErrors(hipFree(d_IMAGE_sig));
  checkCudaErrors(hipFree(d_CONV_amp));
  checkCudaErrors(hipFree(d_CONV_mu));
  checkCudaErrors(hipFree(d_CONV_sig));
  checkCudaErrors(hipFree(d_CONV_CONV_amp));
  checkCudaErrors(hipFree(d_CONV_CONV_mu));
  checkCudaErrors(hipFree(d_CONV_CONV_sig));
}



template <typename T> 
void prepare_for_convolution(T* d_IMAGE, T* d_IMAGE_ext, int dim_x, int dim_y){
    dim3 Dg_2D, Db_2D;
    Db_2D.x = BLOCK_SIZE_L2_X;
    Db_2D.y = BLOCK_SIZE_L2_Y;
    Db_2D.z = 1;
    Dg_2D.x = ceil((dim_x+4)/T(BLOCK_SIZE_L2_X));
    Dg_2D.y = ceil((dim_y+4)/T(BLOCK_SIZE_L2_Y));
    Dg_2D.z = 1;
  	checkCudaErrors(hipMemset(d_IMAGE_ext, 0., (dim_x+4)*(dim_y+4)*sizeof(T)));
//    checkCudaErrors(hipDeviceSynchronize());
//    init_extended_array_sort<T><<<Dg_2D,Db_2D>>>(d_IMAGE, d_IMAGE_ext, dim_x, dim_y);
    checkCudaErrors(hipDeviceSynchronize());
    perform_mirror_effect_before_convolution<T><<<Dg_2D,Db_2D>>>(d_IMAGE, d_IMAGE_ext, dim_x, dim_y);
}

template <typename T> void conv_twice_and_copy(T* d_IMAGE_amp, T* d_conv_amp, T* d_conv_conv_amp, int image_x, int image_y, float* temps)
{
    hipEvent_t record_event[7];
    float tmp_temps[6] = {0.,0.,0.,0.,0.,0.};
    for (int i=0;i<7;i++){
        checkCudaErrors(hipEventCreate(record_event+i));   
    }

    // Record the start event
    dim3 Dg_2D_EXT, Db_2D_EXT;
    Db_2D_EXT.x = BLOCK_SIZE_L2_X;
    Db_2D_EXT.y = BLOCK_SIZE_L2_Y;
    Db_2D_EXT.z = 1;
    Dg_2D_EXT.x = ceil((image_x+4)/T(BLOCK_SIZE_L2_X));
    Dg_2D_EXT.y = ceil((image_y+4)/T(BLOCK_SIZE_L2_Y));
    Dg_2D_EXT.z = 1;

    dim3 Dg_2D, Db_2D;
    Db_2D.x = BLOCK_SIZE_L2_X;
    Db_2D.y = BLOCK_SIZE_L2_Y;
    Db_2D.z = 1;
    Dg_2D.x = ceil((image_x)/T(BLOCK_SIZE_L2_X));
    Dg_2D.y = ceil((image_y)/T(BLOCK_SIZE_L2_Y));
    Dg_2D.z = 1;

    int size_i = (image_x+4)  * (image_y+4)  * sizeof(T);

      checkCudaErrors(hipDeviceSynchronize());
      checkCudaErrors(hipEventRecord(record_event[0], NULL));
      checkCudaErrors(hipDeviceSynchronize());

    T* d_RESULTAT_first_conv;
    hipMalloc((void**)&d_RESULTAT_first_conv, size_i);
  	checkCudaErrors(hipMemset(d_RESULTAT_first_conv, 0., size_i));
    T* d_RESULTAT_second_conv;
    hipMalloc((void**)&d_RESULTAT_second_conv, size_i);
	  checkCudaErrors(hipMemset(d_RESULTAT_second_conv, 0., size_i));

    T* d_IMAGE_amp_ext = NULL;
    checkCudaErrors(hipMalloc(&d_IMAGE_amp_ext, size_i));
    checkCudaErrors(hipMemset(d_IMAGE_amp_ext, 0., size_i));

    prepare_for_convolution<T>(d_IMAGE_amp, d_IMAGE_amp_ext, image_x, image_y);

      checkCudaErrors(hipDeviceSynchronize());
      checkCudaErrors(hipEventSynchronize(record_event[1]));    
      checkCudaErrors(hipEventRecord(record_event[1], NULL));
      checkCudaErrors(hipDeviceSynchronize());

    ConvKernel<T><<<Dg_2D_EXT, Db_2D_EXT>>>(d_RESULTAT_first_conv,  d_IMAGE_amp_ext, image_x+4, image_y+4);

      checkCudaErrors(hipDeviceSynchronize());
      checkCudaErrors(hipEventRecord(record_event[2], NULL));
      checkCudaErrors(hipEventSynchronize(record_event[2]));
      checkCudaErrors(hipDeviceSynchronize());

    copy_gpu<T><<<Dg_2D, Db_2D>>>(d_conv_amp, d_RESULTAT_first_conv, image_x, image_y);

      checkCudaErrors(hipDeviceSynchronize());
      checkCudaErrors(hipEventRecord(record_event[3], NULL));
      checkCudaErrors(hipEventSynchronize(record_event[3]));
      checkCudaErrors(hipDeviceSynchronize());

    prepare_for_convolution<T>(d_conv_amp, d_RESULTAT_first_conv, image_x, image_y);

      checkCudaErrors(hipDeviceSynchronize());
      checkCudaErrors(hipEventRecord(record_event[4], NULL));
      checkCudaErrors(hipEventSynchronize(record_event[4]));
      checkCudaErrors(hipDeviceSynchronize());
 
    ConvKernel<T><<<Dg_2D_EXT, Db_2D_EXT>>>(d_RESULTAT_second_conv,  d_RESULTAT_first_conv, image_x+4, image_y+4);

      checkCudaErrors(hipDeviceSynchronize());
      checkCudaErrors(hipEventRecord(record_event[5], NULL));
      checkCudaErrors(hipEventSynchronize(record_event[5]));
      checkCudaErrors(hipDeviceSynchronize());

    copy_gpu<T><<<Dg_2D, Db_2D>>>(d_conv_conv_amp, d_RESULTAT_second_conv, image_x, image_y);

      checkCudaErrors(hipDeviceSynchronize());
      checkCudaErrors(hipEventRecord(record_event[6], NULL));
      checkCudaErrors(hipEventSynchronize(record_event[6]));
      checkCudaErrors(hipDeviceSynchronize());

    for(int i = 0; i<7-1; i++){
      checkCudaErrors(hipEventElapsedTime(tmp_temps+i, record_event[i], record_event[i+1]));
    }

    temps[0] += tmp_temps[0];
    temps[1] += tmp_temps[1];
    temps[2] += tmp_temps[2];
    temps[3] += tmp_temps[3];
    temps[4] += tmp_temps[4];
    temps[5] += tmp_temps[5];

    checkCudaErrors(hipFree(d_IMAGE_amp_ext));
    checkCudaErrors(hipFree(d_RESULTAT_first_conv));
    checkCudaErrors(hipFree(d_RESULTAT_second_conv));

}

template <typename T> void update_array_f_dev_sort(T lambda, T lambda_var, T* array_f_dev, T* map_image_dev, T* map_conv_dev, int indice_x, int indice_y, int k, T* b_params){

    T* array_f_dev_bis = NULL;
    hipMalloc((void**)&array_f_dev_bis, 1*sizeof(T));
	  checkCudaErrors(hipMemset(array_f_dev_bis, 0., 1*sizeof(T)));

    int size_j = (indice_x)  * (indice_y)  * sizeof(T);
    T* map_norm_dev = NULL;
    hipMalloc((void**)&map_norm_dev, size_j);
  	checkCudaErrors(hipMemset(map_norm_dev, 0., indice_x*indice_y*sizeof(T)));

    dim3 Dg_L2, Db_L2;

    Db_L2.x = BLOCK_SIZE_L2_X;
    Db_L2.y = BLOCK_SIZE_L2_Y;
    Db_L2.z = 1;

    Dg_L2.x = ceil(T(indice_x)/T(BLOCK_SIZE_L2_X));
    Dg_L2.y = ceil(T(indice_y)/T(BLOCK_SIZE_L2_Y));
    Dg_L2.z = 1;

    kernel_norm_map_simple_sort<T><<<Dg_L2,Db_L2>>>(lambda, lambda_var, map_norm_dev, map_conv_dev, map_image_dev, indice_x, indice_y, k, b_params);

    checkCudaErrors(hipDeviceSynchronize());

    reduction_loop<T>(map_norm_dev, array_f_dev_bis, indice_x*indice_y);

    checkCudaErrors(hipDeviceSynchronize());

    add_first_elements_sort<T><<<1,1>>>(array_f_dev_bis, array_f_dev);

    checkCudaErrors(hipDeviceSynchronize());

    checkCudaErrors(hipFree(map_norm_dev));
    checkCudaErrors(hipFree(array_f_dev_bis));
}

template <typename T> void update_array_f_dev_sort(T lambda, T* array_f_dev, T* map_dev, int indice_x, int indice_y){

    T* array_f_dev_bis = NULL;
    hipMalloc((void**)&array_f_dev_bis, 1*sizeof(T));
  	checkCudaErrors(hipMemset(array_f_dev_bis, 0., 1*sizeof(array_f_dev_bis[0])));

    unsigned long int size_j = (indice_x)  * (indice_y)  * sizeof(T);
    T* map_norm_dev = NULL;
    hipMalloc((void**)&map_norm_dev, size_j);
	  checkCudaErrors(hipMemset(map_norm_dev, 0., indice_x*indice_y*sizeof(map_norm_dev[0])));

    dim3 Dg_L2, Db_L2;

    Db_L2.x = BLOCK_SIZE_L2_X;
    Db_L2.y = BLOCK_SIZE_L2_Y;
    Db_L2.z = 1;

    Dg_L2.x = ceil(T(indice_x)/T(BLOCK_SIZE_L2_X));
    Dg_L2.y = ceil(T(indice_y)/T(BLOCK_SIZE_L2_Y));
    Dg_L2.z = 1;

    kernel_norm_map_simple_sort<T><<<Dg_L2,Db_L2>>>(lambda, map_norm_dev, map_dev, indice_x, indice_y);

    checkCudaErrors(hipDeviceSynchronize());

    reduction_loop<T>(map_norm_dev, array_f_dev_bis, indice_x*indice_y);

    checkCudaErrors(hipDeviceSynchronize());

    add_first_elements_sort<T><<<1,1>>>(array_f_dev_bis, array_f_dev);

    checkCudaErrors(hipDeviceSynchronize());

    checkCudaErrors(hipFree(map_norm_dev));
    checkCudaErrors(hipFree(array_f_dev_bis));
}
    
template <typename T> void update_array_f_dev_sort_fast(T lambda_amp, T lambda_mu, T lambda_sig, T lambda_var_sig, T* array_f_dev, T* map_conv_amp_dev, T* map_conv_mu_dev, T* map_conv_sig_dev, T* map_image_sig_dev, int indice_x, int indice_y, int k, T* b_params_dev, float* temps){

    hipEvent_t record_event[3];
    float time_msec[2] = {0.,0.};
    for (int i=0;i<3;i++){
        checkCudaErrors(hipEventCreate(record_event+i));   
    }
    bool print = false;

    // Record the start event
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipEventRecord(record_event[0], NULL));

    T* array_f_dev_bis = NULL;
    hipMalloc((void**)&array_f_dev_bis, 1*sizeof(T));
	checkCudaErrors(hipMemset(array_f_dev_bis, 0., 1*sizeof(T)));

    int size_j = (indice_x)  * (indice_y)  * sizeof(T);
    T* map_norm_dev = NULL;
    hipMalloc((void**)&map_norm_dev, size_j);
	checkCudaErrors(hipMemset(map_norm_dev, 0., indice_x*indice_y*sizeof(T)));

    dim3 Dg_L2, Db_L2;

    Db_L2.x = BLOCK_SIZE_L2_X;
    Db_L2.y = BLOCK_SIZE_L2_Y;
    Db_L2.z = 1;

    Dg_L2.x = ceil(T(indice_x)/T(BLOCK_SIZE_L2_X));
    Dg_L2.y = ceil(T(indice_y)/T(BLOCK_SIZE_L2_Y));
    Dg_L2.z = 1;

    compute_R_map<T><<<Dg_L2,Db_L2>>>(lambda_amp, lambda_mu, lambda_sig, lambda_var_sig, map_norm_dev, map_conv_amp_dev, map_conv_mu_dev, map_conv_sig_dev, map_image_sig_dev, indice_x, indice_y, k, b_params_dev);

    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipEventSynchronize(record_event[1]));    
    checkCudaErrors(hipEventRecord(record_event[1], NULL));
    checkCudaErrors(hipDeviceSynchronize());

/*
    printf("--> Début print un morceau de map_conv_dev :\n");
    checkCudaErrors(hipDeviceSynchronize());
    display_dev_complete_sort<T><<<1,1>>>(map_conv_dev,4);//indice_x*indice_y);
    checkCudaErrors(hipDeviceSynchronize());
    printf("--> Fin print un morceau de map_conv_dev :\n");
    checkCudaErrors(hipDeviceSynchronize());
*/
//    printf("Dg_ud = %d, %d, %d ; Db_ud = %d, %d, %d\n",Dg_ud.x,Dg_ud.y,Dg_ud.z,Db_ud.x,Db_ud.y,Db_ud.z);

    checkCudaErrors(hipDeviceSynchronize());

    reduction_loop<T>(map_norm_dev, array_f_dev_bis, indice_x*indice_y);

    checkCudaErrors(hipDeviceSynchronize());

    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipEventRecord(record_event[2], NULL));
    checkCudaErrors(hipEventSynchronize(record_event[2]));
    checkCudaErrors(hipDeviceSynchronize());


/*
    if(indice_x>=256 && print){
        checkCudaErrors(hipDeviceSynchronize());
        printf("Début f convolution :\n");
        checkCudaErrors(hipDeviceSynchronize());
        display_dev_sort<T><<<1,1>>>(array_f_dev_bis);
        display_dev_complete_sort<T><<<1,1>>>(map_conv_amp_dev,15);
        checkCudaErrors(hipDeviceSynchronize());
        printf("Fin f convolution\n");
        checkCudaErrors(hipDeviceSynchronize());
        std::cin.ignore();
    }
*/

    add_first_elements_sort<T><<<1,1>>>(array_f_dev_bis, array_f_dev);

    checkCudaErrors(hipDeviceSynchronize());

    for(int i = 0; i<3-1; i++){
      checkCudaErrors(hipEventElapsedTime(time_msec+i, record_event[i], record_event[i+1]));
    }

    temps[0]+=time_msec[0];
    temps[1]+=time_msec[1];
 
    
    checkCudaErrors(hipFree(map_norm_dev));
    checkCudaErrors(hipFree(array_f_dev_bis));
}

template double compute_residual_and_f_less_memory<double>(double*, int*, int, double*, int*, int, double*, int*, int, double*, int*, int, int, int, int, int,float*);
template double compute_residual_and_f<double>(double*, int*, int, double*, int*, int, double*, int*, int, double*, int*, int, int, int, int, int);
template void gradient_L_2_beta<double>(double*, int*, int, double*, int*, int, double*, int*, int, double*, int*, int, int, float*);
template void reduction_loop<double>(double*, double*, int);
template void regularization<double>(double*, double*, double*, double&, int, int, int, parameters<double>&, float*);
template void prepare_for_convolution<double>(double*, double*, int, int);
template void conv_twice_and_copy<double>(double*, double*, double*, int, int, float*);
template void update_array_f_dev_sort<double>(double, double, double*, double*, double*, int, int, int, double*);
template void update_array_f_dev_sort<double>(double, double*, double*, int, int);
template void update_array_f_dev_sort_fast<double>(double, double, double, double, double*, double*, double*, double*, double*, int, int, int, double*, float*);

template float compute_residual_and_f_less_memory<float>(float*, int*, int, float*, int*, int, float*, int*, int, float*, int*, int, int, int, int, int, float*);
template float compute_residual_and_f<float>(float*, int*, int, float*, int*, int, float*, int*, int, float*, int*, int, int, int, int, int);
template void gradient_L_2_beta<float>(float*, int*, int, float*, int*, int, float*, int*, int, float*, int*, int, int, float*);
template void reduction_loop<float>(float*, float*, int);
template void regularization<float>(float*, float*, float*, float&, int, int, int, parameters<float>&, float*);
template void prepare_for_convolution<float>(float*, float*, int, int);
template void conv_twice_and_copy<float>(float*, float*, float*, int, int, float*);
template void update_array_f_dev_sort<float>(float, float, float*, float*, float*, int, int, int, float*);
template void update_array_f_dev_sort<float>(float, float*, float*, int, int);
template void update_array_f_dev_sort_fast<float>(float, float, float, float, float*, float*, float*, float*, float*, int, int, int, float*, float*);
