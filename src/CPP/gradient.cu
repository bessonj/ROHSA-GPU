#include <iostream> 
#include <stdio.h>
#include <stdlib.h>
//#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include "gradient.hpp"
#include "model.hpp"

#include "kernel_gradient.cuh"





void gradient(double* dF_over_dB, int* taille_dF_over_dB, int product_taille_dF_over_dB, double* params, int* taille_params, int product_taille_params, model &M)
{
   printf("test début gradient %d\n", dF_over_dB[0]);

   double* dF_over_dB_dev;
   double* params_dev;
   int* taille_dF_over_dB_dev;
   int* taille_params_dev;

   checkCudaErrors(hipMalloc((void**)&dF_over_dB_dev, product_taille_dF_over_dB*sizeof(double)));
   checkCudaErrors(hipMalloc((void**)&params_dev, product_taille_params*sizeof(double)));
   checkCudaErrors(hipMalloc((void**)&taille_dF_over_dB_dev, 4*sizeof(int)));
   checkCudaErrors(hipMalloc((void**)&taille_params_dev, 4*sizeof(int)));

   checkCudaErrors(hipMemcpy(dF_over_dB_dev, dF_over_dB, product_taille_dF_over_dB*sizeof(double), hipMemcpyHostToDevice));
   checkCudaErrors(hipMemcpy(params_dev, params, product_taille_params*sizeof(double), hipMemcpyHostToDevice));
   checkCudaErrors(hipMemcpy(taille_dF_over_dB_dev, taille_dF_over_dB, 4*sizeof(int), hipMemcpyHostToDevice));
   checkCudaErrors(hipMemcpy(taille_params_dev, taille_params, 4*sizeof(int), hipMemcpyHostToDevice));

//   hipMalloc((void**)&dF_over_dB_dev, product_taille_dF_over_dB*sizeof(double));

//   hipMemcpy(dF_over_dB_dev, dF_over_dB, product_taille_dF_over_dB*sizeof(double), hipMemcpyHostToDevice);


   dim3 Db, Dg;

/*
     int N = taille_dF_over_dB[3]*taille_dF_over_dB[2]*taille_dF_over_dB[1]*taille_dF_over_dB[0];

      Db.x = BLOCK_SIZE_X;
      Db.y = BLOCK_SIZE_Y;
      Db.z = BLOCK_SIZE_Z;

      if (N%BLOCK_SIZE_X == 0 && N%BLOCK_SIZE_Y == 0 && N%BLOCK_SIZE_Z == 0)
      {
          Dg.x = taille_dF_over_dB[3]/BLOCK_SIZE_X;
          Dg.y = taille_dF_over_dB[2]/BLOCK_SIZE_Y;
          Dg.z = taille_dF_over_dB[1]/BLOCK_SIZE_Z;
      }
      else
      {
          Dg.x = taille_dF_over_dB[3]/BLOCK_SIZE_X+1;
          Dg.y = taille_dF_over_dB[2]/BLOCK_SIZE_Y+1;
          Dg.z = taille_dF_over_dB[1]/BLOCK_SIZE_Z+1;
      }
      gradient_kernel_0<<< Dg , Db >>>(dF_over_dB_dev, taille_dF_over_dB_dev, params_dev, taille_params_dev, M.n_gauss);
*/

      int N = taille_dF_over_dB[3]*taille_dF_over_dB[2];
      Db.x = BLOCK_SIZE_X;
      Db.y = 1;
      Db.z = 1;

      if (N%BLOCK_SIZE_X == 0)
      {
          Dg.x = N/BLOCK_SIZE_X;
          Dg.y = 1;
          Dg.z = 1;
      }
      else
      {
          Dg.x = N/BLOCK_SIZE_X+1;
          Dg.y = 1;
          Dg.z = 1;
      }


//      gradient_kernel_test<<< Dg , Db >>>(dF_over_dB_dev, taille_dF_over_dB_dev, params_dev, taille_params_dev, M.n_gauss);
      gradient_kernel_1<<< Dg , Db >>>(dF_over_dB_dev, taille_dF_over_dB_dev, params_dev, taille_params_dev, M.n_gauss);
//      gradient_kernel_test<<< Dg , Db >>>(dF_over_dB_dev, taille_dF_over_dB_dev, params_dev, taille_params_dev, M.n_gauss);


  checkCudaErrors(hipMemcpy(dF_over_dB, dF_over_dB_dev, product_taille_dF_over_dB*sizeof(double), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(params, params_dev, product_taille_params*sizeof(double), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(taille_dF_over_dB, taille_dF_over_dB_dev, 4*sizeof(int), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(taille_params, taille_params_dev, 4*sizeof(int), hipMemcpyDeviceToHost));
  // hipMemcpy(dF_over_dB, dF_over_dB_dev, product_taille_dF_over_dB*sizeof(double), hipMemcpyDeviceToHost);

  checkCudaErrors(hipFree(dF_over_dB_dev));
  checkCudaErrors(hipFree(taille_dF_over_dB_dev));
  checkCudaErrors(hipFree(params_dev));
  checkCudaErrors(hipFree(taille_params_dev));

   for(int p; p<2000; p++)
     {
	std::cout<<"dF_over_dB["<<p<<"] = "<<dF_over_dB[p]<<std::endl;
//        printf("p =  %d et dF_over_dB = %f\n",p,dF_over_dB[p]);
     }
   for(int p; p<10; p++)
     {
        printf("p =  %d et taille_dF_over_dB = %d\n",p,taille_dF_over_dB[p]);
     }
   for(int p; p<10; p++)
     {
        printf("p =  %d et params = %f\n",p,params[p]);
     }
   for(int p; p<10; p++)
     {
        printf("p =  %d et taille_params = %d\n",p,taille_params[p]);
     }

     printf("SIZE = %d\n",Dg.x);

}

